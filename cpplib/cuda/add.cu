/*
 * use CUDA C++ compiler to compile
 * nvcc add.cu -o add_cuda
 *
 * profile: nvprof ./add_cuda
 * on Tesla P40 
 * one thread add call takes 201.23ms
 * one thread block 1024 threads takes 3.6280ms
 * many thread blocks takes 2.9605ms 
 *
 * see help https://devblogs.nvidia.com/even-easier-introduction-cuda/
*/

#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

#define THREADS 1024
#define BLOCKS(N) (N + THREADS - 1) / THREADS

// CUDA Kernel function to add the elements of two arrays on the GPU
__global__ void add(int n, float *x, float *y)
{
  // the following two lines is just for 1 block many threads
  //int index = threadIdx.x;
  //int stride = blockDim.x;
  // for many blocks and many threads
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
      y[i] = x[i] + y[i];
}


int main(void)
{
  int N = 1<<20; // 1M elements

  // Allocate Unified Memory -- accessible from CPU or GPU
  float *x, *y;
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Run kernel on 1M elements on the GPU
  // launch one GPU thread
  // add<<<1, 1>>>(N, x, y);
  // launch 1024 threads
  // add<<<1, THREADS>>>(N, x, y);
  // launch many blocks for each block have 1024 threads
  add<<<BLOCKS(N), THREADS>>>(N, x, y);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  hipFree(x);
  hipFree(y);

  return 0;
}
