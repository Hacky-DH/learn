/*
 * get infomation of GPU
 * nvcc info.cu -o info
*/

#include <hip/hip_runtime.h>
#include <cstdio>

int main() {
    int dev = 0;
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, dev);
    printf("GPU: %s\n", prop.name);
    printf("SM num: %d\n", prop.multiProcessorCount);
    printf("max threads per thread block: %d\n", prop.maxThreadsPerBlock);
    printf("max threads per SM: %d\n", prop.maxThreadsPerMultiProcessor);
    printf("total memory: %.2f GB\n", prop.totalGlobalMem/1024./1024/1024);
    printf("shared memory per SM: %.2f KB\n", prop.sharedMemPerMultiprocessor/1024.);
    printf("shared memory per thread block: %.2f KB\n", prop.sharedMemPerBlock/1024.);
    printf("max threads dim: %d,%d,%d\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    return 0;
}
